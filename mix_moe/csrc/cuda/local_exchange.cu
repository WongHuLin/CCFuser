#include "hip/hip_runtime.h"
#include "local_exchange.cuh"
#include "utils.h"
#include <torch/extension.h>


__global__
void assign_pos_kernel(int* cum_count, const long* gate, long* pos,
        size_t numel, size_t topk) {
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < numel) {
        long gate_idx = gate[idx];
        if (gate_idx > -1) {
            int p = atomicSub(cum_count + gate_idx, 1);
            pos[p - 1] = (long)idx;
        }
    }
}

void fmoe_cuda_assign_pos_impl(
        int* cum_count, const long* gate, long* pos,
        const size_t batch_size, const size_t topk,
        CudaStreamManager* smgr) {
    size_t numel = batch_size * topk;
    assign_pos_kernel
        <<<CEIL(numel, 256), 256, 0, smgr->torchStream()>>>
        (cum_count, gate, pos, numel, topk);
}

#define PERTHREAD_EXPERTS 256

#ifdef FMOE_USE_HIP
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif

__global__
void expert_count_kernel(const long* gate_idx, int* expert_count,
        const size_t batch_size, const size_t n_expert) {
    int res_tmp[PERTHREAD_EXPERTS] = {0};
    long expert_min = blockIdx.x * PERTHREAD_EXPERTS;
    long expert_max = expert_min + PERTHREAD_EXPERTS;
    if (expert_max > n_expert) {
        expert_max = n_expert;
    }
    for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
        long idx = gate_idx[i];
        if (idx == -1) {
            continue;
        }
        if (idx < expert_min || idx >= expert_max) {
            continue;
        }
        res_tmp[idx - expert_min] += 1;
    }
    for (int i = expert_min; i < expert_max; ++i) {
        int x = res_tmp[i - expert_min];
#pragma unroll
        for (int j = 1; j < WARP_SIZE; j <<= 1) {
#ifdef FMOE_USE_HIP
            x = x + __shfl_down(x, j);
#else
            x = x + __shfl_down_sync(-1u, x, j);
#endif
        }
        if (threadIdx.x % WARP_SIZE == 0) {
            atomicAdd(expert_count + i, x);
        }
    }
}

void fmoe_cuda_expert_count_impl(
        const long* gate_idx, int* expert_count,
        const size_t batch_size, const size_t n_expert,
        CudaStreamManager* smgr) {
    expert_count_kernel
        <<<CEIL(n_expert, PERTHREAD_EXPERTS), 256, 0, smgr->torchStream()>>>
        (gate_idx, expert_count, batch_size, n_expert);
}

void _assign_pos(
    torch::Tensor cum_count,
    torch::Tensor gate,
    torch::Tensor pos) {
    auto smgr = getCudaStreamManager(cum_count.device().index());
    auto gate_shp = gate.sizes();
    size_t batch_size = gate_shp[0], topk = 1;
    if (gate_shp.size() == 2) {
        topk = gate_shp[1];
    }
    fmoe_cuda_assign_pos_impl(
            cum_count.data_ptr<int>(),
            gate.data_ptr<long>(),
            pos.data_ptr<long>(),
            batch_size, topk, smgr);
}

void _expert_count(
        torch::Tensor gate_idx,
        torch::Tensor expert_count) {
    auto smgr = getCudaStreamManager(gate_idx.device().index());
    auto batch_size = gate_idx.numel();
    auto n_expert = expert_count.numel();
    fmoe_cuda_expert_count_impl(
            gate_idx.data_ptr<long>(),
            expert_count.data_ptr<int>(),
            batch_size, n_expert, smgr);
}
